#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <vector>
#include <algorithm>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <string>
#include <time.h>

#include <cusp/krylov/gmres.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/cg.h>
#include <cusp/monitor.h>
#include <cusp/print.h>

#include <cusp/csr_matrix.h>
#include <cusp/array1d.h>



#include "gpu_timer.h"

using namespace std;

void print_matrix(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp){
	for (int i = 0; i < h_A_cusp.num_rows; ++i)
	{
		int j = 0;
		int k = h_A_cusp.row_offsets[i];
		// printf("%2d-> ", i);
		while(j < h_A_cusp.num_rows && k < h_A_cusp.row_offsets[i+1]){
			if(h_A_cusp.column_indices[k] == j){
				if(h_A_cusp.values[k] < 10)
					cout << h_A_cusp.values[k] << "  ";
				else
					cout << h_A_cusp.values[k] << " ";

				k++;
			}else{
				cout << "0" << "  ";
			}

			j++;
		}

		while(j < h_A_cusp.num_rows){
			cout << "0" << "  ";
			j++;
		}

		cout << endl;
	}
	cout << endl;
}


void fill_matrix_using_junction(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp,
								float* h_b,
								float* h_left, float* h_principal, float* h_right,
								int rows,
								float* gi, vector<vector<int> > &junction_list, float* junction_sums,
								int &tridiag_nnz, int &offdiag_nnz){

	tridiag_nnz = 0;
	offdiag_nnz = 0;

	int DEBUG = 0;

	if(DEBUG){
		// Printing junction list
		for (int i = 0; i < rows; ++i)
		{
			cout << i << " -> " ;
			for (int j = 0; j < junction_list[i].size(); ++j)
			{
				cout <<  junction_list[i][j] << " ";
			}
			cout << endl;
		}	
	}
	

	// Generating symmetrix admittance graph
	// using junction information.
	vector<pair<int,float> > non_zero_elements;
	int node1,node2;
	float admittance;
	float junction_sum;

	for (int i = 0; i < rows; ++i)
	{
		// Main diagonal element
		admittance = gi[i];
		non_zero_elements.push_back(make_pair(i*rows+i, admittance));

		tridiag_nnz++; // [i,i] is part of tri diagonal

		junction_sum = junction_sums[i];
		for (int j = 0; j < junction_list[i].size(); ++j)
		{	
			node1 = junction_list[i][j];
			for (int k = j+1; k < junction_list[i].size(); ++k)
			{
				node2 = junction_list[i][k];
				admittance = (float)(gi[node1]*gi[node2])/junction_sum;

				
				//cout << junction_sum << " " << gi[node1] << " " << gi[node2] << " " << admittance << endl;

				// Pushing element and its symmetry.
				non_zero_elements.push_back(make_pair(node1*rows+node2, admittance));
				non_zero_elements.push_back(make_pair(node2*rows+node1, admittance));

				if(abs(node2-node1) == 1) // left or right principal diagonal
					tridiag_nnz += 2;
				else
					offdiag_nnz += 2;
			}
		}
	}


	// Initializing a cusp csr matrix
	int nnz = non_zero_elements.size();
	h_A_cusp.resize(rows, rows, nnz);

	// Getting elements in csr format.
	// and populating tri diagonal
	sort(non_zero_elements.begin(), non_zero_elements.end());

	int r,c,value;
	for (int i = 0; i < nnz; ++i)
	{
		r = non_zero_elements[i].first/rows;
		c = non_zero_elements[i].first%rows;
		value = non_zero_elements[i].second;

		h_A_cusp.row_offsets[r]++;
		h_A_cusp.column_indices[i] = c;
		h_A_cusp.values[i] = value;

		if(r==c)
			h_principal[r] = value;
		if(r==c+1)
			h_left[r] = value;
		if(r==c-1)
			h_right[r] = value;
	}

	int temp;
	int sum = 0;
	// Scan operation on rowPtr;
	for (int i = 0; i < rows+1; ++i)
	{
		temp = h_A_cusp.row_offsets[i];
		h_A_cusp.row_offsets[i] = sum;
		sum += temp;
	}

	//cusp::print(h_A_cusp);
	if(DEBUG)
		print_matrix(h_A_cusp);


	// Populating rhs
	for (int i = 0; i < rows; ++i)
		//h_b[i] = (rand()%20)+2;
		h_b[i] = 1;

}

void generate_neural_structure(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp,
								float* h_b,
								float* h_left, float* h_principal, float* h_right,
								int rows, int num_mutations,
								int &tridiag_nnz, int &offdiag_nnz){

	// Juntion list with compartment numbers.
	vector< vector<int> > junction_list(rows);
	float rand_number;

	float* gi = new float[rows];
	float* junction_sums = new float[rows];

	// Generating random admittances
	// and initializing junctions
	for (int i = 0; i < rows; ++i){
		rand_number = (rand()%20) + 2;
		gi[i] = rand_number;

		junction_sums[i] = gi[i];
		junction_list[i].push_back(i);
	}


	// Marking random components for mutations.
	bool* mutations = new bool[rows];
	int mutations_found = 0;
	int mutated_comp;

	while(mutations_found != num_mutations){
		mutated_comp = (rand()%(rows-2))+2;

		if(!mutations[mutated_comp]){
			mutations_found++;
			mutations[mutated_comp] = true;
		}
	}

	// Assimilating junction information.
	int prev_node;
	for (int i = 1; i < rows; ++i)
	{
		if(!mutations[i]){
			junction_list[i-1].push_back(i);
			junction_sums[i-1] += gi[i];
		}else{
			// Find the component to associate 
			prev_node = rand()%(i-1);
			//cout << "(" << prev_node << "," << i << ")" << endl;

			junction_list[prev_node].push_back(i);
			junction_sums[prev_node] += gi[i];
		}
	}

	fill_matrix_using_junction(h_A_cusp, h_b,
								h_left, h_principal, h_right,
								rows,
								gi, junction_list, junction_sums,
								tridiag_nnz, offdiag_nnz);


}

void read_neuron_structure(char* file_name, cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp,
								float* h_b,
								float* h_left, float* h_principal, float* h_right,
								int rows, int &tridiag_nnz, int &offdiag_nnz){

	// Juntion list with compartment numbers.
	vector< vector<int> > junction_list(rows);
	float rand_number;

	float* gi = new float[rows];
	float* junction_sums = new float[rows];

	// Generating random admittances
	// and initializing junctions
	for (int i = 0; i < rows; ++i){
		rand_number = (rand()%20) + 2;
		gi[i] = (float)rand_number;

		junction_sums[i] = gi[i];
		junction_list[i].push_back(i);
	}

	// read from file and get junction data.
	ifstream input_file(file_name);
	string line;

	int start,end;
	int parent,child;
	float temp_float;
	while(getline(input_file, line)){
		if(line[0] != '#'){
			stringstream ss(line);
			ss >> child;
			ss >> parent;
			ss >> temp_float;
			ss >> temp_float;
			ss >> temp_float;
			ss >> temp_float;
			ss >> parent;

			if(parent != -1){
				junction_list[parent-1].push_back(child-1);
				junction_sums[parent-1] += gi[child-1];
			}

			//cout << child << " " << parent << endl;
		}
	}

	input_file.close();

	fill_matrix_using_junction(h_A_cusp, h_b,
								h_left, h_principal, h_right,
								rows,
								gi, junction_list, junction_sums,
								tridiag_nnz, offdiag_nnz);	
}


int get_rows_from_file(char* file_name){
	ifstream input_file(file_name);
	string line;

	int rows = 0;
	while(getline(input_file, line)){
		if(line[0] != '#')
			rows++;
	}

	input_file.ignore();
	input_file.close();


	return rows;

}


int main(int argc, char *argv[])
{	

	bool ANALYSIS = false;
	bool MAKE_DIAG_DOMINANT = false;
	bool FROM_FILE = true;

	int rows, num_mutations;

	int tridiag_nnz = 0;
	int offdiag_nnz = 0;

	// set stopping criteria:
    int  iteration_limit    = 500;
    float  relative_tolerance = 1e-6;

	GpuTimer tridiagTimer;
	GpuTimer cuspZeroTimer;
	GpuTimer cuspHintTimer;

    // cusparse handle
    hipsparseHandle_t cusparse_handle = 0;
    hipsparseCreate(&cusparse_handle);

	srand(time(NULL));

	FROM_FILE = atoi(argv[1]);

	if(FROM_FILE){

		rows = get_rows_from_file(argv[2]);

		if(argc > 3)
			ANALYSIS = atoi(argv[3]);

		if(argc > 4)
			iteration_limit = atoi(argv[4]);

		if(argc > 5)
			relative_tolerance = pow(10,-1*atoi(argv[5]));

	}else{
		rows = atoi(argv[2]);
		//int num_mutations = atoi(argv[2];
		num_mutations = (atof(argv[3])*(rows-2))/100;

		if(argc > 4)
			ANALYSIS = atoi(argv[4]);

		if(argc > 5)
			iteration_limit = atoi(argv[5]);

		if(argc > 6)
			relative_tolerance = pow(10,-1*atoi(argv[6]));

	}

	// Matrix details
	cusp::csr_matrix<int, float, cusp::host_memory> h_A;

	// Pointers for tri-diagonal in host and device
	float* h_left, *h_principal, *h_right;
	float* d_left, *d_principal, *d_right;

	float* h_b;
	float* d_b;

	// Allocating memory for tri diagonal
	h_left = (float*) calloc(rows, sizeof(float));
	h_principal = (float*) calloc(rows, sizeof(float));
	h_right = (float*) calloc(rows, sizeof(float));

	hipMalloc((void**)&d_left, rows*sizeof(float));
	hipMalloc((void**)&d_principal, rows*sizeof(float));
	hipMalloc((void**)&d_right, rows*sizeof(float));

	// Allocating memory for rhs
	h_b = (float*) calloc(rows+1, sizeof(float));
	hipMalloc((void**)& d_b, rows*sizeof(float));

	if(FROM_FILE){
		read_neuron_structure(argv[2] ,h_A, h_b ,
							  h_left, h_principal, h_right,
							  rows, tridiag_nnz, offdiag_nnz);
	}else{
		generate_neural_structure(h_A, h_b,
								h_left, h_principal, h_right,
								rows, num_mutations, tridiag_nnz, offdiag_nnz);	
	}
	


	// Copy to gpu
    hipMemcpy(d_left, h_left, sizeof(float)*rows, hipMemcpyHostToDevice);
    hipMemcpy(d_principal, h_principal, sizeof(float)*rows, hipMemcpyHostToDevice);
    hipMemcpy(d_right, h_right, sizeof(float)*rows, hipMemcpyHostToDevice);

    hipMemcpy(d_b, h_b, sizeof(float)*rows, hipMemcpyHostToDevice);

    // Generate device csr matrix
	cusp::csr_matrix<int, float, cusp::device_memory> d_A(h_A);

	// Solving Tri-diag(A)
    tridiagTimer.Start();
    cusparseSgtsv(cusparse_handle,
    			  h_A.num_rows,
    			  1,
    			  d_left, d_principal, d_right,
    			  d_b, h_A.num_rows);
    hipDeviceSynchronize();
    tridiagTimer.Stop();

    float* h_tr_sol = (float*) calloc(h_A.num_rows, sizeof(float));
    hipMemcpy(h_tr_sol, d_b, sizeof(float)* h_A.num_rows, hipMemcpyDeviceToHost);

    // Solving CG using tridiag

	// allocate storage for solution (x) and right hand side (b)
	cusp::array1d<float, cusp::host_memory> cusp_h_x(h_A.num_rows);
	cusp::array1d<float, cusp::host_memory> cusp_h_b(h_A.num_rows);

	for (int i = 0; i < h_A.num_rows; ++i)
	{
		cusp_h_x[i] = h_tr_sol[i];
		cusp_h_b[i] = h_b[i];
	}

	//cusp::print(cusp_h_x);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> cusp_d_clever_x(cusp_h_x);
    cusp::array1d<float, cusp::device_memory> cusp_d_zero_x(h_A.num_rows, 0);

    cusp::array1d<float, cusp::device_memory> cusp_d_clever_b(cusp_h_b);
    cusp::array1d<float, cusp::device_memory> cusp_d_zero_b(cusp_h_b);


    cusp::monitor<float> cleverMonitor(cusp_d_clever_b, iteration_limit, relative_tolerance, 0, !ANALYSIS);
    cusp::monitor<float> zeroMonitor(cusp_d_zero_b, iteration_limit, relative_tolerance, 0, !ANALYSIS);

    // solve the linear system A * x = b with the Conjugate Gradient method
    cuspZeroTimer.Start();
        cusp::krylov::gmres(d_A, cusp_d_zero_x, cusp_d_zero_b, iteration_limit, zeroMonitor);
        //cusp::krylov::cg(d_A, cusp_d_zero_x, cusp_d_zero_b, zeroMonitor);
        hipDeviceSynchronize();
    cuspZeroTimer.Stop();


    cuspHintTimer.Start();
    	cusp::krylov::gmres(d_A, cusp_d_clever_x, cusp_d_clever_b, iteration_limit, cleverMonitor);
        //cusp::krylov::cg(d_A, cusp_d_clever_x, cusp_d_clever_b, cleverMonitor);
    	hipDeviceSynchronize();
    cuspHintTimer.Stop();
    

    float tridiagTime = tridiagTimer.Elapsed();
    float cuspHintTime = cuspHintTimer.Elapsed();
    float cuspZeroTime = cuspZeroTimer.Elapsed();

    float clever_time = tridiagTime+cuspHintTime;
    float speedup = cuspZeroTime/clever_time;
    int clever_iterations = cleverMonitor.iteration_count();
    int bench_iterations = zeroMonitor.iteration_count();

    float offdiag_perc = ((float)offdiag_nnz*100)/h_A.num_entries;
    float tridiag_occupancy = ((float)tridiag_nnz*100)/(3*rows-2);


    if(ANALYSIS){
    	string fn(argv[2]);
    	if(FROM_FILE){
    		int index = fn.size()-1;
	    	while(fn[index] != '/'){
	    		index--;
	    	}
	    	fn = fn.substr(index+1, fn.size()-1-index);
    	}

    	cout << "$" << fn << " " << rows << " " << h_A.num_entries << " " << tridiag_nnz << " " << offdiag_nnz << " " << offdiag_perc << " " << tridiag_occupancy << endl;
        cout << "#" << fn << " " << speedup << " " << clever_time << " " << cuspZeroTime << " " << clever_iterations << " " << bench_iterations << " " << tridiagTime << " " << cuspHintTime << endl;
    }else{
        printf("Speedup %.2f\n",speedup);
        printf("Clever Time %.2f %d (%.2f + %.2f)\n", clever_time, clever_iterations, tridiagTime, cuspHintTime);
        printf("Bench  Time %.2f %d \n", cuspZeroTime, bench_iterations);
		printf("Number of coponenets : %d\n", rows);
		printf("Number of nonzero el : %d\n", h_A.num_entries);
		printf("Number of tridiag el : %d\n", tridiag_nnz);
		printf("Number of offdiag el : %d\n", offdiag_nnz);
		printf("Off-diagonal %%       : %.2f\n", offdiag_perc);
		printf("Tridiag-Occupancy    : %.2f\n", tridiag_occupancy);
	}        
    

	return 0;
}
