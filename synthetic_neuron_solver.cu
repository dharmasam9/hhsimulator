#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <vector>
#include <algorithm>
#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cusp/krylov/gmres.h>
#include <cusp/monitor.h>
#include <cusp/print.h>

#include <cusp/csr_matrix.h>
#include <cusp/array1d.h>

#include <iostream>

#include "gpu_timer.h"

using namespace std;

void print_matrix(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp){
	for (int i = 0; i < h_A_cusp.num_rows; ++i)
	{
		int j = 0;
		int k = h_A_cusp.row_offsets[i];
		printf("%2d-> ", i);
		while(j < h_A_cusp.num_rows && k < h_A_cusp.row_offsets[i+1]){
			if(h_A_cusp.column_indices[k] == j){
				if(h_A_cusp.values[k] < 10)
					cout << "0" << h_A_cusp.values[k] << " ";
				else
					cout << h_A_cusp.values[k] << " ";
				k++;
			}else{
				cout << "00" << " ";
			}

			j++;
		}
		cout << endl;
	}
	cout << endl;
}


void generate_neural_structure(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp,
								float* h_b,
								float* h_left, float* h_principal, float* h_right,
								int rows, int num_mutations){

	// Juntion list with compartment numbers.
	vector< vector<int> > junction_list(rows);
	float rand_number;

	float* gi = new float[rows];
	float* junction_sums = new float[rows];

	// Generating random admittances
	// and initializing junctions
	for (int i = 0; i < rows; ++i){
		rand_number = rand()%20 + 2;
		gi[i] = rand_number;

		junction_sums[i] += gi[i];
		junction_list[i].push_back(i);
	}


	// Marking random components for mutations.
	bool* mutations = new bool[rows];
	int mutations_found = 0;
	int mutated_comp;

	while(mutations_found != num_mutations){
		mutated_comp = (rand()%(rows-2))+2;

		if(!mutations[mutated_comp]){
			mutations_found++;
			mutations[mutated_comp] = true;
		}
	}

	// Assimilating junction information.
	int prev_node;
	for (int i = 1; i < rows; ++i)
	{
		if(!mutations[i]){
			junction_list[i-1].push_back(i);
			junction_sums[i-1] += gi[i];
		}else{
			// Find the component to associate 
			prev_node = rand()%i;
			// cout << "(" << prev_node << "," << i << ")" << endl;

			junction_list[prev_node].push_back(i);
			junction_sums[prev_node] += gi[i];
		}
	}

	/*
	// Printing junction list
	for (int i = 0; i < rows; ++i)
	{
		cout << i << " -> " ;
		for (int j = 0; j < junction_list[i].size(); ++j)
		{
			cout <<  junction_list[i][j] << " ";
		}
		cout << endl;
	}
	*/


	// Generating symmetrix admittance graph
	// using junction information.
	vector<pair<int,float> > non_zero_elements;
	int node1,node2;
	float admittance;
	float junction_sum;

	for (int i = 0; i < rows; ++i)
	{
		// Main diagonal element
		admittance = gi[i];
		non_zero_elements.push_back(make_pair(i*rows+i, admittance));

		junction_sum = junction_sums[i];
		for (int j = 0; j < junction_list[i].size(); ++j)
		{	
			node1 = junction_list[i][j];
			for (int k = j+1; k < junction_list[i].size(); ++k)
			{
				node2 = junction_list[i][k];
				admittance = (gi[node1]*gi[node2])/junction_sum;

				// Pushing element and its symmetry.
				non_zero_elements.push_back(make_pair(node1*rows+node2, admittance));
				non_zero_elements.push_back(make_pair(node2*rows+node1, admittance));
			}
		}
	}


	// Initializing a cusp csr matrix
	int nnz = non_zero_elements.size();
	h_A_cusp.resize(rows, rows, nnz);

	// Getting elements in csr format.
	// and populating tri diagonal
	sort(non_zero_elements.begin(), non_zero_elements.end());

	int r,c,value;
	for (int i = 0; i < nnz; ++i)
	{
		r = non_zero_elements[i].first/rows;
		c = non_zero_elements[i].first%rows;
		value = non_zero_elements[i].second;

		h_A_cusp.row_offsets[r]++;
		h_A_cusp.column_indices[i] = c;
		h_A_cusp.values[i] = value;

		if(r==c)
			h_principal[r] = value;
		if(r==c+1)
			h_left[r] = value;
		if(r==c-1)
			h_right[r] = value;
	}

	int temp;
	int sum = 0;
	// Scan operation on rowPtr;
	for (int i = 0; i < rows+1; ++i)
	{
		temp = h_A_cusp.row_offsets[i];
		h_A_cusp.row_offsets[i] = sum;
		sum += temp;
	}

	//cusp::print(h_A_cusp);
	//print_matrix(h_A_cusp);


	// Populating rhs
	for (int i = 0; i < rows; ++i)
		h_b[i] = 1;
	

}


int main(int argc, char const *argv[])
{	

	bool ANALYSIS = false;
	bool MAKE_DIAG_DOMINANT = false;

	// set stopping criteria:
    int  iteration_limit    = 500;
    float  relative_tolerance = 1e-3;

	GpuTimer tridiagTimer;
	GpuTimer cuspZeroTimer;
	GpuTimer cuspHintTimer;

    // cusparse handle
    hipsparseHandle_t cusparse_handle = 0;
    hipsparseCreate(&cusparse_handle);

	srand(time(NULL));

	int rows = atoi(argv[1]);
	//int num_mutations = atoi(argv[2];
	int num_mutations = (atoi(argv[2])*(rows-2))/100;

	if(argc > 3)
		iteration_limit = atoi(argv[3]);

	if(argc > 4)
		relative_tolerance = pow(10,-1*atoi(argv[4]));


	// Matrix details
	cusp::csr_matrix<int, float, cusp::host_memory> h_A;

	// Pointers for tri-diagonal in host and device
	float* h_left, *h_principal, *h_right;
	float* d_left, *d_principal, *d_right;

	float* h_b;
	float* d_b;

	// Allocating memory for tri diagonal
	h_left = (float*) calloc(rows, sizeof(float));
	h_principal = (float*) calloc(rows, sizeof(float));
	h_right = (float*) calloc(rows, sizeof(float));

	hipMalloc((void**)&d_left, rows*sizeof(float));
	hipMalloc((void**)&d_principal, rows*sizeof(float));
	hipMalloc((void**)&d_right, rows*sizeof(float));

	// Allocating memory for rhs
	h_b = (float*) calloc(rows+1, sizeof(float));
	hipMalloc((void**)& d_b, rows*sizeof(float));

	generate_neural_structure(h_A, h_b,
								h_left, h_principal, h_right,
								rows, num_mutations);


	// Copy to gpu
    hipMemcpy(d_left, h_left, sizeof(float)*rows, hipMemcpyHostToDevice);
    hipMemcpy(d_principal, h_principal, sizeof(float)*rows, hipMemcpyHostToDevice);
    hipMemcpy(d_right, h_right, sizeof(float)*rows, hipMemcpyHostToDevice);

    hipMemcpy(d_b, h_b, sizeof(float)*rows, hipMemcpyHostToDevice);

    // Generate device csr matrix
	cusp::csr_matrix<int, float, cusp::device_memory> d_A(h_A);

	// Solving Tri-diag(A)
    tridiagTimer.Start();
    cusparseSgtsv(cusparse_handle,
    			  h_A.num_rows,
    			  1,
    			  d_left, d_principal, d_right,
    			  d_b, h_A.num_rows);
    hipDeviceSynchronize();
    tridiagTimer.Stop();

    float* h_tr_sol = (float*) calloc(h_A.num_rows, sizeof(float));
    hipMemcpy(h_tr_sol, d_b, sizeof(float)* h_A.num_rows, hipMemcpyDeviceToHost);

    // Solving CG using tridiag

	// allocate storage for solution (x) and right hand side (b)
	cusp::array1d<float, cusp::host_memory> cusp_h_x(h_A.num_rows);
	cusp::array1d<float, cusp::host_memory> cusp_h_b(h_A.num_rows);

	for (int i = 0; i < h_A.num_rows; ++i)
	{
		cusp_h_x[i] = h_tr_sol[i];
		cusp_h_b[i] = h_b[i];
	}

	//cusp::print(cusp_h_x);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> cusp_d_clever_x(cusp_h_x);
    cusp::array1d<float, cusp::device_memory> cusp_d_zero_x(h_A.num_rows, 0);

    cusp::array1d<float, cusp::device_memory> cusp_d_clever_b(cusp_h_b);
    cusp::array1d<float, cusp::device_memory> cusp_d_zero_b(cusp_h_b);


    cusp::monitor<float> cleverMonitor(cusp_d_clever_b, iteration_limit, relative_tolerance, 0, !ANALYSIS);
    cusp::monitor<float> zeroMonitor(cusp_d_zero_b, iteration_limit, relative_tolerance, 0, !ANALYSIS);

    // solve the linear system A * x = b with the Conjugate Gradient method
    cuspZeroTimer.Start();
        cusp::krylov::gmres(d_A, cusp_d_zero_x, cusp_d_zero_b, iteration_limit, zeroMonitor);
        //cusp::krylov::bicgstab(d_A, cusp_d_zero_x, cusp_d_zero_b, zeroMonitor);
        hipDeviceSynchronize();
    cuspZeroTimer.Stop();

    cuspHintTimer.Start();
    	cusp::krylov::gmres(d_A, cusp_d_clever_x, cusp_d_clever_b, iteration_limit, cleverMonitor);
        //cusp::krylov::bicgstab(d_A, cusp_d_clever_x, cusp_d_clever_b, cleverMonitor);
    	hipDeviceSynchronize();
    cuspHintTimer.Stop();


    float tridiagTime = tridiagTimer.Elapsed();
    float cuspHintTime = cuspHintTimer.Elapsed();
    float cuspZeroTime = cuspZeroTimer.Elapsed();

    float clever_time = tridiagTime+cuspHintTime;
    float speedup = cuspZeroTime/clever_time;
    int clever_iterations = cleverMonitor.iteration_count();
    int bench_iterations = zeroMonitor.iteration_count();


    if(ANALYSIS){
        cout << speedup << " " << clever_time << " " << cuspZeroTime << " " << clever_iterations << " " << bench_iterations << " " << tridiagTime << " " << cuspHintTime << endl;
    }else{
        cout <<  speedup << endl;
        cout << "My time: " << tridiagTime+cuspHintTime << " (" << tridiagTime << "," << cuspHintTime << ")" << " in iterations " << cleverMonitor.iteration_count() <<  endl;
        cout << "Bnc tym: " << cuspZeroTime << " in iterations " << zeroMonitor.iteration_count() << endl;
	}        
    

	return 0;
}