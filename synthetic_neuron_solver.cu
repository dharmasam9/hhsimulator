#include <hip/hip_runtime.h>

#include <vector>
#include <algorithm>
#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cusp/krylov/gmres.h>
#include <cusp/monitor.h>
#include <cusp/print.h>

#include <cusp/csr_matrix.h>
#include <cusp/array1d.h>

#include <iostream>

#include "gpu_timer.h"

using namespace std;

void print_matrix(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp){
	for (int i = 0; i < h_A_cusp.num_rows; ++i)
	{
		int j = 0;
		int k = h_A_cusp.row_offsets[i];
		printf("%2d-> ", i);
		while(j < h_A_cusp.num_rows && k < h_A_cusp.row_offsets[i+1]){
			if(h_A_cusp.column_indices[k] == j){
				if(h_A_cusp.values[k] < 10)
					cout << "0" << h_A_cusp.values[k] << " ";
				else
					cout << h_A_cusp.values[k] << " ";
				k++;
			}else{
				cout << "00" << " ";
			}

			j++;
		}
		cout << endl;
	}
	cout << endl;
}


void generate_neural_structure(cusp::csr_matrix<int, float, cusp::host_memory> &h_A_cusp,
								cusp::array1d<float,cusp::host_memory> &h_b_cusp,
								float* h_left, float* h_principal, float* h_right,
								int rows, int num_mutations){

	// Juntion list with compartment numbers.
	vector< vector<int> > junction_list(rows);
	float rand_number;

	float* gi = new float[rows];
	float* junction_sums = new float[rows];

	// Generating random admittances
	// and initializing junctions
	for (int i = 0; i < rows; ++i){
		rand_number = rand()%20 + 2;
		gi[i] = rand_number;

		junction_sums[i] += gi[i];
		junction_list[i].push_back(i);
	}


	// Marking random components for mutations.
	bool* mutations = new bool[rows];
	int mutations_found = 0;
	int mutated_comp;

	while(mutations_found != num_mutations){
		mutated_comp = (rand()%(rows-2))+2;

		if(!mutations[mutated_comp]){
			mutations_found++;
			mutations[mutated_comp] = true;
		}
	}

	// Assimilating junction information.
	int prev_node;
	for (int i = 1; i < rows; ++i)
	{
		if(!mutations[i]){
			junction_list[i-1].push_back(i);
			junction_sums[i-1] += gi[i];
		}else{
			// Find the component to associate 
			prev_node = rand()%i;
			// cout << "(" << prev_node << "," << i << ")" << endl;

			junction_list[prev_node].push_back(i);
			junction_sums[prev_node] += gi[i];
		}
	}

	/*
	// Printing junction list
	for (int i = 0; i < rows; ++i)
	{
		cout << i << " -> " ;
		for (int j = 0; j < junction_list[i].size(); ++j)
		{
			cout <<  junction_list[i][j] << " ";
		}
		cout << endl;
	}
	*/


	// Generating symmetrix admittance graph
	// using junction information.
	vector<pair<int,float> > non_zero_elements;
	int node1,node2;
	float admittance;
	float junction_sum;

	for (int i = 0; i < rows; ++i)
	{
		// Main diagonal element
		admittance = gi[i];
		non_zero_elements.push_back(make_pair(i*rows+i, admittance));

		junction_sum = junction_sums[i];
		for (int j = 0; j < junction_list[i].size(); ++j)
		{	
			node1 = junction_list[i][j];
			for (int k = j+1; k < junction_list[i].size(); ++k)
			{
				node2 = junction_list[i][k];
				admittance = (gi[node1]*gi[node2])/junction_sum;

				// Pushing element and its symmetry.
				non_zero_elements.push_back(make_pair(node1*rows+node2, admittance));
				non_zero_elements.push_back(make_pair(node2*rows+node1, admittance));
			}
		}
	}


	// Initializing a cusp csr matrix
	int nnz = non_zero_elements.size();
	h_A_cusp.resize(rows, rows, nnz);

	// Getting elements in csr format.
	// and populating tri diagonal
	sort(non_zero_elements.begin(), non_zero_elements.end());

	int r,c,value;
	for (int i = 0; i < nnz; ++i)
	{
		r = non_zero_elements[i].first/rows;
		c = non_zero_elements[i].first%rows;
		value = non_zero_elements[i].second;

		h_A_cusp.row_offsets[r]++;
		h_A_cusp.column_indices[i] = c;
		h_A_cusp.values[i] = value;

		if(r==c)
			h_principal[r] = value;
		if(r==c+1)
			h_left[r] = value;
		if(r==c-1)
			h_right[r] = value;
	}

	int temp;
	int sum = 0;
	// Scan operation on rowPtr;
	for (int i = 0; i < rows+1; ++i)
	{
		temp = h_A_cusp.row_offsets[i];
		h_A_cusp.row_offsets[i] = sum;
		sum += temp;
	}

	//cusp::print(h_A_cusp);
	//print_matrix(h_A_cusp);



	/*
	// Populating rhs
	for (int i = 0; i < rows; ++i)
		h_b_cusp[i] = 1;
	*/

}


int main(int argc, char const *argv[])
{	

	srand(time(NULL));

	int rows = atoi(argv[1]);
	// int num_mutations = (atoi(argv[2])*(rows-2))/100;
	int num_mutations = atoi(argv[2]);

	// Matrix details
	cusp::csr_matrix<int, float, cusp::host_memory> h_A_cusp;
	cusp::array1d<float,cusp::host_memory> h_b_cusp(rows);

	// Pointers for tri-diagonal in host and device
	float* h_left, *h_principal, *h_right;
	float* d_left, *d_principal, *d_right;

	// Allocating memory for tri diagonal
	h_left = (float*) calloc(rows, sizeof(float));
	h_principal = (float*) calloc(rows, sizeof(float));
	h_right = (float*) calloc(rows, sizeof(float));

	generate_neural_structure(h_A_cusp, h_b_cusp,
								h_left, h_principal, h_right,
								rows, num_mutations);

	return 0;
}