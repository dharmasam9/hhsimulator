#include "hip/hip_runtime.h"

#include "gpu_hsolve_utils.cu"


using namespace std;

int main(int argc, char *argv[])
{

	bool ANALYSIS = true;
	bool DEBUG = false;
	bool FROM_FILE = false;

	int tridiag_nnz = 0;
	int offdiag_nnz = 0;

	// set stopping criteria:
	int  iteration_limit    = 50;
	float  relative_tolerance = 1e-6;

	// cusparse handle
	hipsparseHandle_t cusparse_handle = 0;
	hipsparseCreate(&cusparse_handle);

	/* initialize random seed: */
	srand (time(NULL));

	// Required arrays
	double simulation_time = 0.3, dT = 0.1;
	int time_steps = 0;

	vector<vector<int> > junction_list;

	int num_comp = 0;
	int* h_channel_counts; // 3*i+k element means ith componenent k=0->Na k=1->K k=2->cl

	double* h_V,*h_Cm, *h_Ga, *h_Rm, *h_Em;
	double* h_gate_m,*h_gate_h,*h_gate_n;
	double* h_current_inj;

	// If from file read the structure or generate structure
	if(argc > 1)
		FROM_FILE = atoi(argv[1]);

	// Setting number of components
	if(FROM_FILE){
		num_comp = get_rows_from_file(argv[2]);

		if(argc > 3)
			simulation_time = atof(argv[3]);

		if(argc > 4)
			dT = atof(argv[4]);

		if(argc > 5)
			DEBUG = atoi(argv[5]);

		if(argc > 6)
			iteration_limit = atoi(argv[6]);

		if(argc > 7)
			relative_tolerance = pow(10,-1*atoi(argv[7]));

		junction_list.resize(num_comp);
		get_structure_from_neuron(argv[2], num_comp, junction_list);

	}else{
		int num_mutations = 1;
		num_comp = 5;

		if(argc > 2)
			num_comp = atoi(argv[2]);		

		if(argc > 3)
			num_mutations = (atof(argv[3])*(num_comp-2))/100; // Branching percentage

		if(argc > 4)
			simulation_time = atof(argv[4]);

		if(argc > 5)
			dT = atof(argv[5]);

		if(argc > 6)
			DEBUG = atoi(argv[6]);

		if(argc > 7)
			iteration_limit = atoi(argv[7]);

		if(argc > 8)
			relative_tolerance = pow(10,-1*atoi(argv[8]));

		junction_list.resize(num_comp);
		generate_random_neuron(num_comp, num_mutations, junction_list);
	}

	// Calculating time_steps
	time_steps = simulation_time/dT;

	// Allocating memory
	h_V  = new double[num_comp]();
	h_Cm = new double[num_comp]();
	h_Ga = new double[num_comp]();
	h_Rm = new double[num_comp]();
	h_Em = new double[num_comp]();

	h_gate_m = new double[num_comp]();
	h_gate_n = new double[num_comp]();
	h_gate_h = new double[num_comp]();

	h_current_inj = new double[time_steps]();
	h_channel_counts = new int[3*num_comp]();


	// Initializing m,h,h
	initialize_gates(num_comp, h_gate_m, h_gate_n, h_gate_h);

	// Full current through out.
	for (int i = 0; i < time_steps; ++i){
		if(i<=100)
			h_current_inj[i] = I_EXT;
		else
			h_current_inj[i] = 0;
	}



	/* Managing current
	int temp = (25*time_steps)/100;
	for (int i = 0; i < temp; ++i){
		h_current_inj[i] = I_EXT;
		h_current_inj[time_steps-temp-i] = I_EXT;
	}
	*/

	// Setting up channels
	// Randomly assigning channel types for chann in compartment.
	for (int i = 0; i < num_comp; ++i)
	{
		int num_channels = max(3,rand()%MAX_CHAN_PER_COMP);
		// Making sure compartment has atleast one Na,K,Cl channel
		int chan_type, na_count = 1, k_count = 1, cl_count = 1;
		for (int j = 0; j < num_channels-3; ++j)
		{
			chan_type = rand()%3;
			switch(chan_type){
				case 0:
					na_count++;
					break;
				case 1:
					k_count++;
					break;
				case 2:
					cl_count++;
					break;
			}
		}

		h_channel_counts[i*3] = na_count;
		h_channel_counts[i*3+1] = k_count;
		h_channel_counts[i*3+2] = cl_count;
	}


	populate_V(h_V, num_comp);
	populate_Cm(h_Cm, num_comp);
	populate_Ga(h_Ga, num_comp);
	populate_Rm(h_Rm, num_comp);
	populate_Em(h_Em, num_comp);

	// ****************************** SetUp Matrix ************************************
	
	// Passive data cm/dt + 1/Rm + G
	cusp::csr_matrix<int, double, cusp::host_memory> h_A_cusp;
	double* h_b;
	double* h_maindiag_passive;
	double* h_tridiag_data;
	int* h_maindiag_map;

	// Allocating memory
	h_b = new double[num_comp]();
	h_maindiag_passive = new double[num_comp]();
	h_tridiag_data = new double[3*num_comp]();
	h_maindiag_map = new int[num_comp]();

	fill_matrix_using_junction(num_comp, junction_list,
								h_A_cusp, h_b,
								h_maindiag_passive, h_tridiag_data, h_maindiag_map,
								h_Cm, h_Ga, h_Rm, dT,
								tridiag_nnz, offdiag_nnz);
	// **************************** Device memory Allocation ****************************
	double* d_V,*d_Cm, *d_Rm, *d_Em;
	double* d_gate_m,*d_gate_h,*d_gate_n;
	double* d_current_inj;
	int* d_channel_counts;
	cusp::array1d<double,cusp::device_memory> d_b_cusp(num_comp);

	cusp::csr_matrix<int, double, cusp::device_memory> d_A_cusp(h_A_cusp);
	double* d_maindiag_passive, *d_tridiag_data;
	int* d_maindiag_map;

	hipMalloc((void**)&d_V, num_comp*sizeof(double));
	hipMalloc((void**)&d_Cm, num_comp*sizeof(double));
	hipMalloc((void**)&d_Rm, num_comp*sizeof(double));
	hipMalloc((void**)&d_Em, num_comp*sizeof(double));

	hipMalloc((void**)&d_gate_m, num_comp*sizeof(double));
	hipMalloc((void**)&d_gate_h, num_comp*sizeof(double));
	hipMalloc((void**)&d_gate_n, num_comp*sizeof(double));

	hipMalloc((void**)&d_current_inj, time_steps*sizeof(double));
	hipMalloc((void**)&d_channel_counts, (3*num_comp)*sizeof(int));

	hipMalloc((void**)&d_maindiag_passive, num_comp*sizeof(double));
	hipMalloc((void**)&d_tridiag_data, (3*num_comp)*sizeof(double));
	hipMalloc((void**)&d_maindiag_map, num_comp*sizeof(int));

	hipMemcpy(d_V, h_V, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_Cm, h_Cm, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_Rm, h_Rm, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_Em, h_Em, sizeof(double)*num_comp, hipMemcpyHostToDevice);

	hipMemcpy(d_gate_m, h_gate_m, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_gate_h, h_gate_h, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_gate_n, h_gate_n, sizeof(double)*num_comp, hipMemcpyHostToDevice);

	hipMemcpy(d_current_inj, h_current_inj, sizeof(double)*time_steps, hipMemcpyHostToDevice);
	hipMemcpy(d_channel_counts, h_channel_counts, sizeof(int)*(3*num_comp), hipMemcpyHostToDevice);

	hipMemcpy(d_maindiag_passive, h_maindiag_passive, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_tridiag_data, h_tridiag_data, sizeof(double)*(3*num_comp), hipMemcpyHostToDevice);
	hipMemcpy(d_maindiag_map, h_maindiag_map, sizeof(int)*num_comp, hipMemcpyHostToDevice);

	// Extra memory for performing simulation.
	double* d_GkSum, *d_GkEkSum;
	hipMalloc((void**)&d_GkSum, num_comp*sizeof(double));
	hipMemset(d_GkSum,0, sizeof(double) * num_comp);
	hipMalloc((void**)&d_GkEkSum, num_comp*sizeof(double));
	hipMemset(d_GkEkSum,0, sizeof(double) * num_comp);


	// **************************** Simulation begins ************************************

	//print_matrix(h_A_cusp);	
	
	// STATE BEFORE SIMULATION
	/*
	if(DEBUG){
		cusp::print(d_A_cusp);
		hipMemcpy(h_tridiag_data, d_tridiag_data, sizeof(double)*(3*num_comp), hipMemcpyDeviceToHost);

		for (int i = 0; i < num_comp; ++i)
			cout << h_tridiag_data[i] << " " << h_tridiag_data[num_comp+i] << " " << h_tridiag_data[2*num_comp+i] << endl;

		// Printing currents
		cusp::print(d_b_cusp);

		// Each run of simulation
		hipMemcpy(h_V, d_V, sizeof(double)*num_comp, hipMemcpyDeviceToHost);

		for (int i = 0; i < num_comp; ++i)
			cout << h_V[i] << endl;		

		cout << "********************************************" << endl;

	}
	*/

	// ************************************************
	if(!ANALYSIS) cout << "SIMULATION BEGINS" << endl;

	ofstream V_file, solver_file;
	V_file.open("output.csv");
	solver_file.open("solver.csv");

	double offdiag_perc = (offdiag_nnz*100.0)/h_A_cusp.num_entries;
	double tridiag_occupancy = (tridiag_nnz * 100.0)/ (3*h_A_cusp.num_rows);
	solver_file << h_A_cusp.num_rows << " " << h_A_cusp.num_entries << " " << tridiag_nnz << " " << offdiag_nnz << " " << offdiag_perc << " " << tridiag_occupancy << endl;

	double h_Vplot[num_comp];
	double h_Mplot[num_comp];
	double h_Nplot[num_comp];
	double h_Hplot[num_comp];

	int NUM_THREAD_PER_BLOCK = 512;
	int NUM_BLOCKS = ceil((num_comp*1.0)/NUM_THREAD_PER_BLOCK);
	
	for (int i = 0; i < time_steps; ++i)
	{

		// GPU Timers
		GpuTimer tridiagTimer;
		GpuTimer cuspZeroTimer;
		GpuTimer cuspHintTimer;
		
		GpuTimer channelTimer;
		GpuTimer currentTimer;

		channelTimer.Start();
			// ADVANCE m,n,h channels
			advance_channel_m<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, d_V, d_gate_m, dT);
			advance_channel_n<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, d_V, d_gate_n, dT);
			advance_channel_h<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, d_V, d_gate_h, dT);
			hipDeviceSynchronize();
		channelTimer.Stop();
		
		currentTimer.Start();
			// CALCULATE Gk and GkEk values
			calculate_gk_gkek_sum<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, d_V, 
												d_gate_m, d_gate_h, d_gate_n, 
												d_channel_counts, 
												d_GkSum, d_GkEkSum);
			hipDeviceSynchronize();

			// CALCULATE currents
			double externalCurrent = h_current_inj[i];
			calculate_currents<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, d_V, d_Cm, dT, 
								d_Em, d_Rm, 
								d_GkEkSum, externalCurrent, 
								thrust::raw_pointer_cast(&d_b_cusp[0]));
			hipDeviceSynchronize();

			// UPDATE matrix and TRIDIAG
			update_matrix<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, num_comp, d_maindiag_passive, d_GkSum, 
						 d_maindiag_map, 
						 thrust::raw_pointer_cast(&(d_A_cusp.values[0])), d_tridiag_data);

			//hipMemcpy(h_b, d_b, num_comp* sizeof(double), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
		currentTimer.Stop();

		// Printing hines matrix, tridiag and currentVector
		// *********************************
		//cusp::print(d_b_cusp);
		if(DEBUG){
			print_iteration_state(d_A_cusp, d_b_cusp);
		}
		
		// *************************************

		// Cloning b,because tri diagonal solver overrites it with answer
		cusp::array1d<double,cusp::device_memory> d_b_cusp_copy1(d_b_cusp);
		cusp::array1d<double,cusp::device_memory> d_b_cusp_copy2(d_b_cusp);
		cusp::array1d<double, cusp::device_memory> d_x_zero_cusp(num_comp, 0);

		// Solver
		tridiagTimer.Start();
		cusparseDgtsv(cusparse_handle,
					  num_comp,
					  1,
					  d_tridiag_data, &d_tridiag_data[num_comp], &d_tridiag_data[num_comp*2],
					  thrust::raw_pointer_cast(&d_b_cusp[0]), num_comp);
		hipDeviceSynchronize();
		tridiagTimer.Stop();


		cusp::monitor<double> cleverMonitor(d_b_cusp_copy1, iteration_limit, relative_tolerance, 0, !ANALYSIS);
		cusp::monitor<float> zeroMonitor(d_b_cusp_copy2, iteration_limit, relative_tolerance, 0, !ANALYSIS);

		// solve the linear system A * x = b with the Conjugate Gradient method
		cuspZeroTimer.Start();
			cusp::krylov::gmres(d_A_cusp, d_x_zero_cusp, d_b_cusp_copy1, iteration_limit, zeroMonitor);
			//cusp::krylov::cg(d_A_cusp, d_x_zero_cusp, d_b_cusp_copy1, zeroMonitor);
			hipDeviceSynchronize();
		cuspZeroTimer.Stop();

		// solve the linear system A * x = b with the Conjugate Gradient method
		cuspHintTimer.Start();
			cusp::krylov::gmres(d_A_cusp, d_b_cusp, d_b_cusp_copy2, iteration_limit, cleverMonitor);
			//cusp::krylov::cg(d_A_cusp, d_b_cusp, d_b_cusp_copy2, zeroMonitor);
			hipDeviceSynchronize();
		cuspHintTimer.Stop();

		// UPDATE V
		update_V<<<NUM_BLOCKS,NUM_THREAD_PER_BLOCK>>>(num_comp, thrust::raw_pointer_cast(&d_b_cusp[0]), d_V);
		hipDeviceSynchronize();

		// ***************************************

		// Transfer V to cpu for plotting
		hipMemcpy(h_Vplot, d_V, num_comp* sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(h_Mplot, d_gate_m, num_comp* sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(h_Hplot, d_gate_h, num_comp* sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(h_Nplot, d_gate_n, num_comp* sizeof(double), hipMemcpyDeviceToHost);
		
		// Timings
		float tridiagTime = tridiagTimer.Elapsed();	
		float cuspHintTime = cuspHintTimer.Elapsed();
		float cuspZeroTime = cuspZeroTimer.Elapsed();
		float channelTime = channelTimer.Elapsed();
		float currentTime = currentTimer.Elapsed();

		float clever_time = tridiagTime+cuspHintTime;
		float speedup = cuspZeroTime/clever_time;
		int clever_iterations = cleverMonitor.iteration_count();
		int bench_iterations = zeroMonitor.iteration_count();

		float channelPerc = (channelTime * 100)/(channelTime + currentTime + clever_time);
		float currentPerc = (currentTime * 100)/(channelTime + currentTime + clever_time);
		float solverPerc = (clever_time * 100)/(channelTime + currentTime + clever_time);

		if(i<10){
			printf("Speedup %.2f\n",speedup);
			printf("Clever Time %.2f %d (%.2f + %.2f)\n", clever_time, clever_iterations, tridiagTime, cuspHintTime);
			printf("Bench  Time %.2f %d \n", cuspZeroTime, bench_iterations);	
			printf("profil Time %.2f %.2f %.2f\n", channelPerc, currentPerc, solverPerc);
		}
		
		solver_file << i << " " <<  speedup << " " << clever_time << " " << cuspZeroTime << " " << clever_iterations << " " << bench_iterations << " " << tridiagTime << " " << cuspHintTime << " " << channelPerc << " " << currentPerc << " " << solverPerc <<  endl;
		V_file << i*dT << "," << h_Vplot[0] <<  "," << h_Mplot[0] << "," << h_Hplot[0] << "," << h_Nplot[0] << "," << clever_iterations << "," << bench_iterations <<  "," << (bench_iterations-clever_iterations) << "," << speedup << endl;
		//cout << i*dT << "," << h_Vplot[0] << endl;
	}

	V_file.close();
	solver_file.close();

	return 0;
}