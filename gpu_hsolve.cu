#include "hip/hip_runtime.h"

#include "gpu_hsolve_utils.cu"


using namespace std;

int main(int argc, char *argv[])
{
	// Whether input is from file or not
	int FROM_FILE = 0;

	if(argc > 1)
		FROM_FILE = atoi(argv[1]);

	/* initialize random seed: */
	srand (time(NULL));

	// Required arrays
	double simulation_time, dT;
	int time_steps = 0;

	vector<vector<int> > junction_list;

	int num_comp = 0;
	int* h_channel_counts; // 3*i+k element means ith componenent k=0->Na k=1->K k=2->cl

	double* h_V,*h_Cm, *h_Ga, *h_Rm, *h_Em;
	double* h_gate_m,*h_gate_h,*h_gate_n;
	double* h_current_inj;

	// Setting up simulation times
	simulation_time = 100;
	dT = 1;
	time_steps = simulation_time/dT;

	// Setting number of components
	if(FROM_FILE){
		num_comp = get_rows_from_file(argv[2]);
		junction_list.resize(num_comp);

		get_structure_from_neuron(argv[2], num_comp, junction_list);

	}else{
		num_comp = 5;
		junction_list.resize(num_comp);

		int num_mutations = 1;
		generate_random_neuron(num_comp, num_mutations, junction_list);
	}

	/*
	// print junction list
	for (int i = 0; i < junction_list.size(); ++i)
	{	
		cout << i << "-> ";
		for (int j = 0; j < junction_list[i].size(); ++j)
			cout << junction_list[i][j] << " ";
		cout << endl;
	}
	*/
	

	// Allocating memory
	h_V  = new double[num_comp];
	h_Cm = new double[num_comp];
	h_Ga = new double[num_comp];
	h_Rm = new double[num_comp];
	h_Em = new double[num_comp];

	h_gate_m = new double[num_comp];
	h_gate_n = new double[num_comp];
	h_gate_h = new double[num_comp];

	h_current_inj = new double[time_steps];


	//First 25% and last 25% of currents to be set
	int temp = (25*time_steps)/100;
	for (int i = 0; i < temp; ++i){
		h_current_inj[i] = I_EXT;
		h_current_inj[time_steps-i] = I_EXT;
	}


	// Setting up channels
	h_channel_counts = new int[3*num_comp];

	// Randomly assigning channel types for chann in compartment.
	for (int i = 0; i < num_comp; ++i)
	{
		int num_channels = max(3,rand()%MAX_CHAN_PER_COMP);
		// Making sure compartment has atleast one Na,K,Cl channel
		int chan_type, na_count = 1, k_count = 1, cl_count = 1;
		for (int j = 0; j < num_channels-3; ++j)
		{
			chan_type = rand()%3;
			switch(chan_type){
				case 0:
					na_count++;
					break;
				case 1:
					k_count++;
					break;
				case 2:
					cl_count++;
					break;
			}
		}

		h_channel_counts[i*3] = na_count;
		h_channel_counts[i*3+1] = k_count;
		h_channel_counts[i*3+2] = cl_count;
	}


	populate_V(h_V, num_comp);
	populate_Cm(h_Cm, num_comp);
	populate_Ga(h_Ga, num_comp);
	populate_Rm(h_Rm, num_comp);
	populate_Em(h_Em, num_comp);

	// ****************************** SetUp Matrix ************************************
	
	// Passive data cm/dt + 1/Rm + G
	cusp::csr_matrix<int, double, cusp::host_memory> h_A_cusp;
	double* h_b;
	double* h_maindiag_passive;
	double* h_tridiag_data;
	int* h_maindiag_map;
	int tridiag_nnz =0, offdiag_nnz = 0;

	// Allocating memory
	h_b = new double[num_comp];
	h_maindiag_passive = new double[num_comp];
	h_tridiag_data = new double[3*num_comp];
	h_maindiag_map = new int[num_comp];

	fill_matrix_using_junction(num_comp, junction_list,
								h_A_cusp, h_b,
								h_maindiag_passive, h_tridiag_data, h_maindiag_map,
								h_Cm, h_Ga, h_Rm, dT,
								tridiag_nnz, offdiag_nnz);
	// **************************** Device memory Allocation ****************************
	double* d_V,*d_Cm, *d_Rm, *d_Em;
	double* d_gate_m,*d_gate_h,*d_gate_n;
	double* d_current_inj;
	double* d_b;

	int* d_channel_counts;

	h_A_cusp.values[0] = 20;
	cusp::csr_matrix<int, double, cusp::device_memory> d_A_cusp(h_A_cusp);
	double* d_maindiag_passive, *d_tridiag_data;
	int* d_maindiag_map;

	hipMalloc((void**)&d_V, num_comp*sizeof(double));
	hipMalloc((void**)&d_Cm, num_comp*sizeof(double));
	hipMalloc((void**)&d_Rm, num_comp*sizeof(double));
	hipMalloc((void**)&d_Em, num_comp*sizeof(double));

	hipMalloc((void**)&d_gate_m, num_comp*sizeof(double));
	hipMalloc((void**)&d_gate_h, num_comp*sizeof(double));
	hipMalloc((void**)&d_gate_n, num_comp*sizeof(double));

	hipMalloc((void**)&d_current_inj, time_steps*sizeof(double));
	hipMalloc((void**)&d_b, num_comp*sizeof(double));

	hipMalloc((void**)&d_channel_counts, num_comp*sizeof(int));

	hipMalloc((void**)&d_maindiag_passive, num_comp*sizeof(double));
	hipMalloc((void**)&d_tridiag_data, num_comp*sizeof(double));
	hipMalloc((void**)&d_maindiag_map, num_comp*sizeof(int));

	hipMemcpy(d_V, h_V, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_Em, h_Cm, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_Em, h_Rm, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_Em, h_Em, sizeof(double)*num_comp, hipMemcpyHostToDevice);

	hipMemcpy(d_gate_m, h_gate_m, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_gate_h, h_gate_h, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_gate_n, h_gate_n, sizeof(double)*num_comp, hipMemcpyHostToDevice);

	hipMemcpy(d_current_inj, h_current_inj, sizeof(double)*time_steps, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(double)*num_comp, hipMemcpyHostToDevice);

	hipMemcpy(d_channel_counts, h_channel_counts, sizeof(double)*num_comp, hipMemcpyHostToDevice);

	hipMemcpy(d_maindiag_passive, h_maindiag_passive, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_tridiag_data, h_tridiag_data, sizeof(double)*num_comp, hipMemcpyHostToDevice);
	hipMemcpy(d_maindiag_map, h_maindiag_map, sizeof(int)*num_comp, hipMemcpyHostToDevice);

	// Extra memory for performing simulation.
	double* d_GkSum, *d_GkEkSum;
	hipMalloc((void**)&d_GkSum, num_comp*sizeof(double));
	hipMemset(d_GkSum,0, sizeof(double) * num_comp);
	hipMalloc((void**)&d_GkEkSum, num_comp*sizeof(double));
	hipMemset(d_GkEkSum,0, sizeof(double) * num_comp);


	// **************************** Simulation begins ************************************
	for (int i = 0; i < 1; ++i)
	{
		// Each run of simulation

		// Advance m,n,h
		advance_channel_m<<<1,num_comp>>>(d_V, d_gate_m, dT);
		advance_channel_n<<<1,num_comp>>>(d_V, d_gate_h, dT);
		advance_channel_h<<<1,num_comp>>>(d_V, d_gate_n, dT);

		/*
		hipMemcpy(h_gate_m, d_gate_m, sizeof(double)*num_comp, hipMemcpyDeviceToHost);

		for (int j = 0; j < num_comp; ++j)
			cout << h_gate_m[j] << " ";
			cout << endl;
		*/
		
		// Calculate Gk and GkEk values
		calculate_gk_gkek_sum<<<1,num_comp>>>(d_V, 
											d_gate_m, d_gate_h, d_gate_n, 
											d_channel_counts, 
											d_GkSum, d_GkEkSum);
		// update right hand side b
		double externalCurrent = h_current_inj[i];
		calculate_currents<<<1,num_comp>>>(d_V, d_Cm, dT, 
							d_Em, d_Rm, 
							d_GkEkSum, externalCurrent, 
							d_b);


		// update main diagonal values in cusp and tridiagdata
		update_matrix<<<1,num_comp>>>(d_maindiag_passive, d_GkSum, 
					 d_maindiag_map, 
					 thrust::raw_pointer_cast(&(d_A_cusp.values[0])), d_tridiag_data);

		// solver
		// update V


	}
	





	return 0;
}